
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void addArrays(const int* A, const int* B, int* C) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	C[i] = A[i] + B[i];
}

int main(void) {
	// Crea los buffer en el host, para los datos de entrada y salida
	int* inputA = (int*)malloc(sizeof(int) * 1024);
	int* inputB = (int*)malloc(sizeof(int) * 1024);
	int* outputC = (int*)malloc(sizeof(int) * 1024);

	// Crea los buffer en la GPU, para los datos de entrada y salida
	int *gpuBuffer_A, *gpuBuffer_B, *gpuBuffer_C;
	hipMalloc(&gpuBuffer_A, sizeof(int)*1024);
	hipMalloc(&gpuBuffer_B, sizeof(int)*1024);
	hipMalloc(&gpuBuffer_C, sizeof(int) * 1024);

	// Inicializa los buffer del host con los valores de entrada
	for (int i = 0; i < 1024; i++) {
		inputA[i] = i; //0,1,2,...,1023
		inputB[i] = 1023 - i; //1023,1022,...,0
	}

	// Copia los valores de entrada desde los buffer del host a los buffer de la GPU
	hipMemcpy(gpuBuffer_A, inputA, sizeof(int) * 1024, hipMemcpyHostToDevice);
	hipMemcpy(gpuBuffer_B, inputB, sizeof(int) * 1024, hipMemcpyHostToDevice);

	// Ejecuta la kernel en la GPU (4 bloques * 256 hilos = 1024 elementos calculados)
	addArrays<<<4, 256>>>(gpuBuffer_A, gpuBuffer_B, gpuBuffer_C);

	// Recupera el resultado desde la GPU y lo pone en un buffer del host
	hipMemcpy(outputC, gpuBuffer_C, sizeof(int) * 1024, hipMemcpyDeviceToHost);

	// Presenta el resultado
	for (int i = 0; i < 1024; i++) {
		printf("Resultados %d: (%d + %d = %d)\n", i, inputA[i], inputB[i], outputC[i]);
	}

	// Libera los recursos
	hipFree(gpuBuffer_A);
	hipFree(gpuBuffer_B);
	hipFree(gpuBuffer_C);
	free(inputA);
	free(inputB);
	free(outputC);
}
